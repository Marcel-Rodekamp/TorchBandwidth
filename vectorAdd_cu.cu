
#include <hip/hip_runtime.h>
#include<iostream>
#include<chrono>
#include<vector>
#include<algorithm>
#include<cmath>

#include<omp.h>

__global__ void vec_add(const double * x, const double * y, double * out, std::size_t N){
    std::size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < N){
        out[i] = x[i] + y[i];
    }
}

__global__ void initVector(double * x, std::size_t N){
    std::size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < N){
        x[i] = static_cast<double>(i);
    }
}

double mean(std::vector<double> v) {
    double res = 0;

    for(auto e: v){
        res+=e;
    }
    res/=v.size();

    return res;
}

double err(std::vector<double> v) {

    double est = mean(v);

    double res = 0;

    for(auto e: v){
        res+= (e - est)*(e - est) ;
    }
    res/=v.size()-1;

    return std::sqrt(res);
}

int main(){
    // Tensor size
    const std::size_t N = std::pow(2,28);

    // Tensor memory  
    // 1 byte = 9.31×10-10 Gb
    const double mem_GB = N * sizeof(double) * 9.31e-10;
    
    // Statistical power
    const int N_meas = 100;

    // Number of sweeps per measurement
    const int N_sweep = 10;

    // store timing data in here
    std::vector<double> timings(N_meas);

    // Define the tensors
    double * x  = nullptr;
    double * y  = nullptr;
    double * out= nullptr;

    hipMalloc(&x,N * sizeof(double));
    hipMalloc(&y,N * sizeof(double));
    hipMalloc(&out,N * sizeof(double));

    int threadsPerBlock = 256;
    int numBlocks = (N + threadsPerBlock - 1)/threadsPerBlock; 

    initVector<<<numBlocks,threadsPerBlock>>>(x,N);
    initVector<<<numBlocks,threadsPerBlock>>>(y,N);
    initVector<<<numBlocks,threadsPerBlock>>>(out,N);

    double start, end;

    // measure T1+T2 and store into Tout N_meas times
    for(int i = 0; i < N_meas; ++i){
        if (i % 10 == 0){
            std::cout << "Measure ID: " << i << "/" << N_meas << std::endl;
        }
        start = omp_get_wtime();
        for(int j = 0; j < N_sweep; ++j){
            vec_add<<<numBlocks,threadsPerBlock>>>(x,y,out,N);
            hipDeviceSynchronize();
        }
        end = omp_get_wtime();
        
        timings[i] = (end-start)/N_sweep;
    }

    // compute and print statistics    
    double est_timings = mean(timings);
    double err_timings = err(timings);

    // the factor 3 comes from:
    // * factor 2: 2 vector load
    // * factor 1.5: 1 vector store
    // => 1.5 * 2 = 3
    double est_bw = 3*mem_GB / est_timings;
    // just simple gaussian error propagation
    double err_bw = err_timings * 3*mem_GB / (est_timings*est_timings); 

    // N operations in vector add
    double est_flops = N  / est_timings;
    // just simple gaussian error propagation
    double err_flops = err_timings * N / (est_timings*est_timings);
 
    std::cout << "* Memory Footprint   : " << mem_GB << " Gb" << std::endl;
    std::cout << "* Min Execution Time : " << *std::min_element(timings.begin(), timings.end()) << " s" << std::endl;
    std::cout << "* Max Execution Time : " << *std::max_element(timings.begin(), timings.end()) << " s" << std::endl;
    std::cout << "* Mean Execution Time: " << est_timings << " +/- " << err_timings << " s" << std::endl;
    std::cout << "* Mean Bandwidth     : " << est_bw << " +/- " << err_bw << " Gb/s" << std::endl;
    std::cout << "* Mean Flops         : " << est_flops << " +/- " << err_flops << " flops" << std::endl;

    std::cout << std::endl;
#pragma omp parallel
    {
#pragma omp single
    std::cout << "* Number threads     : " << omp_get_num_threads() << "/" << omp_get_max_threads() << std::endl;
    }
    std::cout << "* use GPU            : " << std::boolalpha << true << std::endl;

    hipFree(x);
    hipFree(y);
    hipFree(out);

    return EXIT_SUCCESS;
}
